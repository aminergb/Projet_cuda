#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <hip/device_functions.h>
#include "costum_library/costumlib.h"

__constant__ char Filter[9];

__global__ void BlurOneKernel(uchar* rgb_b, uchar* rgb_g, uchar* rgb_r, uchar* sortie_b, uchar* sortie_g, uchar* sortie_r, std::size_t cols, std::size_t rows, uchar paddingSize) {
    //essayer d'enlever le holder
    // Set the padding size and filter size
    //int S = Ss;
    //int paddingSize = S;
    unsigned int filterSize = (2 * paddingSize) + 1;
    int paddedW = 2 * paddingSize + cols;
    int paddedH = 2 * paddingSize + rows;
   


    //char Filter[9]={1, 1, 1, 1, 1, 1, 1, 1, 1};


    // Set the pixel coordinate

    //bizzare , lorsque je fais j=i+paddingsize (รงa m'affiche noir)
    auto  j = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
    auto  i = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

    // The multiply-add operation for the pixel coordinate ( j, i )
    if (j >= paddingSize && j < paddedW - paddingSize && i >= paddingSize && i < paddedH - paddingSize) {
        auto oPixelPos = (i - paddingSize) * cols + (j - paddingSize);
        sortie_b[oPixelPos] = 0.0;
        sortie_g[oPixelPos] = 0.0;
        sortie_r[oPixelPos] = 0.0;
        auto channelB = 0.0;
        auto  channelG = 0.0;
        auto  channelR = 0.0;

        for (int k = -paddingSize; k <= paddingSize; k++) {

            for (int l = -paddingSize; l <= paddingSize; l++) {
                //le probleme est la 
                auto iPixelPos = (i + k) * paddedW + (j + l);

                auto coefPos = (k + paddingSize) * filterSize + (l + paddingSize);
                channelB += (rgb_b[iPixelPos] * Filter[coefPos]) / 9;
                channelG += (rgb_g[iPixelPos] * Filter[coefPos]) / 9;
                channelR += (rgb_r[iPixelPos] * Filter[coefPos]) / 9;

            }
            //enlever ça et remplacer plus haut
            sortie_b[oPixelPos] = channelB;
            sortie_g[oPixelPos] = channelG;
            sortie_r[oPixelPos] = channelR;

        }

    }

}


int main()
{
    cv::Mat m_in = cv::imread("images/Lenna.jpg", cv::IMREAD_COLOR);
    //creation d'une mat qui contiendra les channels apres traitement 
    //
    cv::Mat mergechannels[3];
    cv::split(m_in, mergechannels);

    auto S = 1;

    auto rgb = m_in.data;
    auto rows = m_in.rows;
    auto cols = m_in.cols;
    auto paddedW = 2 * S + cols;
    auto paddedH = 2 * S + rows;
    std::vector<unsigned char > g(cols * rows);
    std::vector<unsigned char > bor(paddedW * paddedH);
    std::vector<unsigned char > blue(cols * rows);
    std::vector<unsigned char > green(cols * rows);
    std::vector<unsigned char > red(cols * rows);

    //faire en sorte de detecter les types d'images 
    cv::Mat m_out(rows, cols, CV_8UC1, g.data());
    cv::Mat m_bor(rows, cols, CV_8UC1, bor.data());
    cv::Mat m_blue(rows, cols, CV_8UC1, blue.data());
    cv::Mat m_green(rows, cols, CV_8UC1, green.data());
    cv::Mat m_red(rows, cols, CV_8UC1, red.data());


    cv::copyMakeBorder(m_in, m_bor, S, S, S, S, CV_HAL_BORDER_CONSTANT, 0);
    auto rows_bo = m_bor.rows;
    auto cols_bo = m_bor.cols;

    auto rgb_bo = m_bor.data;
    cv::Mat splitchannels[3];

    cv::split(m_bor, splitchannels);
    auto bl = splitchannels[0].data;
    auto gr = splitchannels[1].data;
    auto re = splitchannels[2].data;

    uchar* blue_h;
    uchar* green_h;
    uchar* red_h;

    //result from device
    uchar* blue_d;
    uchar* green_d;
    uchar* red_d;



    char mask_h[9] = { 1, 1, 1, 1, 1, 1, 1, 1, 1 };

    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Filter), mask_h, 9));
    //allocation vers gpu
    dim3 t(32, 32);
    dim3 b((cols_bo - 1) / t.x + 1, (rows_bo - 1) / t.y + 1);
    hipEvent_t start, stop;
    int count = 0;
      
    while (count <= 1000) {
        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, 0));

        HANDLE_ERROR(hipMalloc(&blue_h, rows_bo * cols_bo));
        HANDLE_ERROR(hipMalloc(&green_h, rows_bo * cols_bo));
        HANDLE_ERROR(hipMalloc(&red_h, rows_bo * cols_bo));

        HANDLE_ERROR(hipHostAlloc((void**)&blue_d, rows * cols * sizeof(*blue_d), hipHostMallocDefault));
        HANDLE_ERROR(hipHostAlloc((void**)&green_d, rows * cols * sizeof(*green_d), hipHostMallocDefault));
        HANDLE_ERROR(hipHostAlloc((void**)&red_d, rows * cols * sizeof(*red_d), hipHostMallocDefault));

      
        HANDLE_ERROR(hipMemcpy(blue_h, bl, rows_bo * cols_bo, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(green_h, gr, rows_bo * cols_bo, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(red_h, re, rows_bo * cols_bo, hipMemcpyHostToDevice));
        

        BlurOneKernel << < b, t >> > (blue_h, green_h, red_h, blue_d, green_d, red_d, cols, rows, S);
        //hipDeviceSynchronize();

        HANDLE_ERROR(hipMemcpy(blue.data(), blue_d, rows * cols, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(green.data(), green_d, rows * cols, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(red.data(), red_d, rows * cols, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipFree(blue_h));
        HANDLE_ERROR(hipFree(green_h));
        HANDLE_ERROR(hipFree(red_h));
        mergechannels[0].data = blue.data();
        mergechannels[1].data = green.data();
        mergechannels[2].data = red.data();
        HANDLE_ERROR(hipHostFree(blue_d));
        HANDLE_ERROR(hipHostFree(green_d));
        HANDLE_ERROR(hipHostFree(red_d));

        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
            start, stop));
        printf("%3.1f\n", elapsedTime);
        HANDLE_ERROR(hipEventDestroy(start));
        HANDLE_ERROR(hipEventDestroy(stop));
        count++;
    }
    //merging the channels
    cv::merge(mergechannels, 3, m_out);
    cv::imshow("f", m_out);
    cv::waitKey();
    cv::imwrite("C:/Users/lagab/Pictures/2ou4t23.jpg", m_out);



    return 0;
}
