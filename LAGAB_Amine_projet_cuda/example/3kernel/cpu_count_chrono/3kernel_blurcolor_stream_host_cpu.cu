#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <hip/device_functions.h>
#include "costum_library/costumlib.h"

__constant__ char Filter[9];

__global__ void BlurColorKernel(uchar* rgb, uchar* sortie, std::size_t cols, std::size_t rows, uchar paddingSize) {
    //essayer d'enlever le holder
    // Set the padding size and filter size

    unsigned int filterSize = (2 * paddingSize) + 1;
    int paddedW = 2 * paddingSize + cols;
    int paddedH = 2 * paddingSize + rows;
    //filter testing
    //{1, 1, 1, 1, 1, 1, 1, 1, 1}


    // Set the pixel coordinate

    //bizzare , lorsque je fais j=i+paddingsize (รงa m'affiche noir)
    auto  j = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
    auto  i = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

    // The multiply-add operation for the pixel coordinate ( j, i )
    if (j >= paddingSize && j < paddedW - paddingSize && i >= paddingSize && i < paddedH - paddingSize) {
        auto oPixelPos = (i - paddingSize) * cols + (j - paddingSize);
        sortie[oPixelPos] = 0.0;
        auto channel = 0.0;
        for (int k = -paddingSize; k <= paddingSize; k++) {
            for (int l = -paddingSize; l <= paddingSize; l++) {
                //le probleme est la 
                auto iPixelPos = (i + k) * paddedW + (j + l);
                auto coefPos = (k + paddingSize) * filterSize + (l + paddingSize);
                channel += (rgb[iPixelPos] * Filter[coefPos]) / 9;
            }
            sortie[oPixelPos] = channel;
        }

    }
}


int main()
{
    int count = 0;
    while (count <= 1000) {

        auto startcpu = std::chrono::steady_clock::now();
    cv::Mat m_in = cv::imread("C:/Users/lagab/Pictures/Lenna.jpg", cv::IMREAD_COLOR);
    //creation d'une mat qui contiendra les channels apres traitement 
    //
    cv::Mat mergechannels[3];
    cv::split(m_in, mergechannels);

    auto S = 1;

    auto rgb = m_in.data;
    auto rows = m_in.rows;
    auto cols = m_in.cols;
    auto paddedW = 2 * S + cols;
    auto paddedH = 2 * S + rows;
    std::vector<unsigned char > g(cols * rows);
    std::vector<unsigned char > bor(paddedW * paddedH);
    std::vector<unsigned char > blue(cols * rows);
    std::vector<unsigned char > green(cols * rows);
    std::vector<unsigned char > red(cols * rows);

    //faire en sorte de detecter les types d'images 
    cv::Mat m_out(rows, cols, CV_8UC1, g.data());
    cv::Mat m_bor(rows, cols, CV_8UC1, bor.data());
    cv::Mat m_blue(rows, cols, CV_8UC1, blue.data());
    cv::Mat m_green(rows, cols, CV_8UC1, green.data());
    cv::Mat m_red(rows, cols, CV_8UC1, red.data());


    cv::copyMakeBorder(m_in, m_bor, S, S, S, S, CV_HAL_BORDER_CONSTANT, 0);
    auto rows_bo = m_bor.rows;
    auto cols_bo = m_bor.cols;

    auto rgb_bo = m_bor.data;
    cv::Mat splitchannels[3];

    cv::split(m_bor, splitchannels);
    auto bl = splitchannels[0].data;
    auto gr = splitchannels[1].data;
    auto re = splitchannels[2].data;

    uchar* blue_h;
    uchar* green_h;
    uchar* red_h;

    //result from device
    uchar* blue_d;
    uchar* green_d;
    uchar* red_d;



    char mask_h[9] = { 1, 1, 1, 1, 1, 1, 1, 1, 1 };
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Filter), mask_h, 9));


    //allocation vers gpu
    dim3 t(32, 32);
    dim3 b((cols_bo - 1) / t.x + 1, (rows_bo - 1) / t.y + 1);
    hipEvent_t start, stop;
    hipStream_t streamblue, streamgreen, streamred;
   

        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, 0));
        HANDLE_ERROR(hipStreamCreate(&streamblue));
        HANDLE_ERROR(hipStreamCreate(&streamgreen));
        HANDLE_ERROR(hipStreamCreate(&streamred));
        HANDLE_ERROR(hipMalloc(&blue_h, rows_bo * cols_bo));
        HANDLE_ERROR(hipMalloc(&green_h, rows_bo * cols_bo));
        HANDLE_ERROR(hipMalloc(&red_h, rows_bo * cols_bo));

        HANDLE_ERROR(hipHostAlloc((void**)&blue_d, rows * cols * sizeof(*blue_d), hipHostMallocDefault));
        HANDLE_ERROR(hipHostAlloc((void**)&green_d, rows * cols * sizeof(*green_d), hipHostMallocDefault));
        HANDLE_ERROR(hipHostAlloc((void**)&red_d, rows * cols * sizeof(*red_d), hipHostMallocDefault));


        HANDLE_ERROR(hipMemcpyAsync(blue_h, bl, rows_bo * cols_bo, hipMemcpyHostToDevice, streamblue));
        HANDLE_ERROR(hipMemcpyAsync(green_h, gr, rows_bo * cols_bo, hipMemcpyHostToDevice, streamgreen));
        HANDLE_ERROR(hipMemcpyAsync(red_h, re, rows_bo * cols_bo, hipMemcpyHostToDevice, streamred));

        BlurColorKernel << < b, t, 0, streamblue >> > (blue_h, blue_d, cols, rows, S);
        BlurColorKernel << < b, t, 0, streamgreen >> > (green_h, green_d, cols, rows, S);
        BlurColorKernel << < b, t, 0, streamred >> > (red_h, red_d, cols, rows, S);

        HANDLE_ERROR(hipMemcpyAsync(blue.data(), blue_d, rows * cols, hipMemcpyDeviceToHost, streamblue));
        HANDLE_ERROR(hipMemcpyAsync(green.data(), green_d, rows * cols, hipMemcpyDeviceToHost, streamgreen));
        HANDLE_ERROR(hipMemcpyAsync(red.data(), red_d, rows * cols, hipMemcpyDeviceToHost, streamred));
        HANDLE_ERROR(hipStreamSynchronize(streamblue));
        HANDLE_ERROR(hipStreamSynchronize(streamgreen));
        HANDLE_ERROR(hipStreamSynchronize(streamred));
        HANDLE_ERROR(hipFree(blue_h));
        HANDLE_ERROR(hipFree(green_h));
        HANDLE_ERROR(hipFree(red_h));
        mergechannels[0].data = blue.data();
        mergechannels[1].data = green.data();
        mergechannels[2].data = red.data();
        HANDLE_ERROR(hipHostFree(blue_d));
        HANDLE_ERROR(hipHostFree(green_d));
        HANDLE_ERROR(hipHostFree(red_d));
        HANDLE_ERROR(hipStreamDestroy(streamblue));
        HANDLE_ERROR(hipStreamDestroy(streamgreen));
        HANDLE_ERROR(hipStreamDestroy(streamred));
        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
            start, stop));
      //  printf("%3.1f\n", elapsedTime);
        HANDLE_ERROR(hipEventDestroy(start));
        HANDLE_ERROR(hipEventDestroy(stop));
       
    //merging the channels
    cv::merge(mergechannels, 3, m_out);
    cv::imwrite("C:/Users/lagab/Pictures/2ou4t23.jpg", m_out);
    auto endcpu = std::chrono::steady_clock::now();
    float elapsedTimeCPU = std::chrono::duration_cast<std::chrono::milliseconds>(endcpu - startcpu).count();
    printf("%3.1f\n", elapsedTimeCPU);

    count++;
    }

   // cv::imshow("f", m_out);
    cv::waitKey();
   


    return 0;
}
